#include "hip/hip_runtime.h"
#include <>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include "cuda_canny_edge.cuh"

#define VERBOSE 1
#define MAX_KERNEL_SIZE 128

// Constants for the GPU constant memory
__constant__ struct {
    float kernel[MAX_KERNEL_SIZE];
    float sum;
} d_gaussian_constants;

// Optimized kernel for horizontal Gaussian smoothing using shared memory
__global__ void gaussian_smooth_x_kernel(const unsigned char *d_image, float *d_temp, int rows, int cols, int kernel_radius) {
    
    extern __shared__ unsigned char s_data_x[];

    int r = blockIdx.y * blockDim.y + threadIdx.y;
    int s_r = threadIdx.y;
    int s_width = blockDim.x + 2 * kernel_radius;

    int global_c_start = blockIdx.x * blockDim.x - kernel_radius;
    int elements_per_thread = (s_width + blockDim.x - 1) / blockDim.x;

    for (int i = 0; i < elements_per_thread; ++i) {
        int idx = threadIdx.x + i * blockDim.x;
        if (idx >= s_width) break;

        int load_c = global_c_start + idx;
        int s_index = s_r * s_width + idx;

        if (r < rows) {
            s_data_x[s_index] = (load_c >= 0 && load_c < cols) ? d_image[r * cols + load_c] : 0;
        } else {
            s_data_x[s_index] = 0;
        }
    }

    __syncthreads();

    int c = blockIdx.x * blockDim.x + threadIdx.x;
    if (r < rows && c < cols) {
        float dot = 0.0f;
        #pragma unroll
        for (int i = -kernel_radius; i <= kernel_radius; i++) {
            int s_c = threadIdx.x + kernel_radius + i;
            dot += s_data_x[s_r * s_width + s_c] * d_gaussian_constants.kernel[kernel_radius + i];
        }
        d_temp[r * cols + c] = dot / d_gaussian_constants.sum;
    }
}

// Optimized kernel for vertical Gaussian smoothing using shared memory
__global__ void gaussian_smooth_y_kernel(const float *d_temp, short int *d_smoothed, int rows, int cols, int kernel_radius) {
    
    extern __shared__ float s_data_y[];

    int c = blockIdx.x * blockDim.x + threadIdx.x;
    int s_c = threadIdx.x;
    int s_height = blockDim.y + 2 * kernel_radius;

    int global_r_start = blockIdx.y * blockDim.y - kernel_radius;
    int elements_per_thread = (s_height + blockDim.y - 1) / blockDim.y;

    for (int i = 0; i < elements_per_thread; ++i) {
        int idx = threadIdx.y + i * blockDim.y;
        if (idx >= s_height) break;

        int load_r = global_r_start + idx;
        int s_index = idx * blockDim.x + s_c;

        if (c < cols) {
            s_data_y[s_index] = (load_r >= 0 && load_r < rows) ? d_temp[load_r * cols + c] : 0.0f;
        } else {
            s_data_y[s_index] = 0.0f;
        }
    }

    __syncthreads();

    int r = blockIdx.y * blockDim.y + threadIdx.y;
    if (r < rows && c < cols) {
        float dot = 0.0f;
        #pragma unroll
        for (int i = -kernel_radius; i <= kernel_radius; i++) {
            int s_r = threadIdx.y + kernel_radius + i;
            dot += s_data_y[s_r * blockDim.x + s_c] * d_gaussian_constants.kernel[kernel_radius + i];
        }
        d_smoothed[r * cols + c] = (short int)(dot * 90.0f / d_gaussian_constants.sum + 0.5f);
    }
}

// Function to create Gaussian kernel and call GPU kernels
void cuda_gaussian_smooth(unsigned char *image, int rows, int cols, float sigma, short int **smoothedim)
{
    // Start CPU timer for the entire function
    double total_start_time = get_time_ms();
    
    hipError_t cudaStatus;
    hipEvent_t start, stop;
    double cpu_start, cpu_end;
    float gpu_time = 0.0f;

    // Create CUDA events
    cudaStatus = hipEventCreate(&start);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipEventCreate failed: %s\n", hipGetErrorString(cudaStatus));
        return;
    }
    cudaStatus = hipEventCreate(&stop);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipEventCreate failed: %s\n", hipGetErrorString(cudaStatus));
        hipEventDestroy(start);
        return;
    }

    int windowsize, center;
    float *kernel;

    struct {
        float kernel[MAX_KERNEL_SIZE];
        float sum;
    } h_gaussian_constants;

    // Create the Gaussian kernel
    make_gaussian_kernel(sigma, &kernel, &windowsize);
    if (windowsize > MAX_KERNEL_SIZE) {
        fprintf(stderr, "Error: Kernel size %d exceeds maximum allowed size %d\n", windowsize, MAX_KERNEL_SIZE);
        free(kernel);
        hipEventDestroy(start);
        hipEventDestroy(stop);
        exit(1);
    }
    center = windowsize / 2;

    // Calculate kernel sum
    float kernel_sum = 0.0f;
    for (int i = 0; i < windowsize; i++) {
        kernel_sum += kernel[i];
    }

    memcpy(h_gaussian_constants.kernel, kernel, windowsize * sizeof(float));
    h_gaussian_constants.sum = kernel_sum;

    // Copy kernel struct to constant memory
    hipEventRecord(start);
    cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(d_gaussian_constants), &h_gaussian_constants, sizeof(h_gaussian_constants));
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&gpu_time, start, stop);
    printf("hipMemcpyToSymbol time: %.2f ms\n", gpu_time);
    
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpyToSymbol failed: %s\n", hipGetErrorString(cudaStatus));
        free(kernel);
        hipEventDestroy(start);
        hipEventDestroy(stop);
        exit(1);
    }

    free(kernel);

    size_t imageSize = rows * cols * sizeof(unsigned char);
    size_t tempSize = rows * cols * sizeof(float);
    size_t smoothSize = rows * cols * sizeof(short int);

    unsigned char *d_image;
    float *d_temp;
    short int *d_smoothed;

    // Allocate device memory
    hipEventRecord(start);
    cudaStatus = hipMalloc((void **)&d_image, imageSize);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&gpu_time, start, stop);
    printf("hipMalloc d_image time: %.2f ms\n", gpu_time);
    
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc d_image failed: %s\n", hipGetErrorString(cudaStatus));
        exit(1);
    }

    hipEventRecord(start);
    cudaStatus = hipMalloc((void **)&d_temp, tempSize);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&gpu_time, start, stop);
    printf("hipMalloc d_temp time: %.2f ms\n", gpu_time);
    
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc d_temp failed: %s\n", hipGetErrorString(cudaStatus));
        hipFree(d_image);
        exit(1);
    }

    hipEventRecord(start);
    cudaStatus = hipMalloc((void **)&d_smoothed, smoothSize);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&gpu_time, start, stop);
    printf("hipMalloc d_smoothed time: %.2f ms\n", gpu_time);
    
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc d_smoothed failed: %s\n", hipGetErrorString(cudaStatus));
        hipFree(d_image);
        hipFree(d_temp);
        exit(1);
    }

    // Copy input image to device
    hipEventRecord(start);
    cudaStatus = hipMemcpy(d_image, image, imageSize, hipMemcpyHostToDevice);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&gpu_time, start, stop);
    printf("hipMemcpy to device time: %.2f ms\n", gpu_time);
    
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy d_image failed: %s\n", hipGetErrorString(cudaStatus));
        hipFree(d_image);
        hipFree(d_temp);
        hipFree(d_smoothed);
        exit(1);
    }

    // Set up grid and block dimensions
    dim3 blockDim_x(32, 8);
    dim3 gridDim_x((cols + blockDim_x.x - 1) / blockDim_x.x, (rows + blockDim_x.y - 1) / blockDim_x.y);

    dim3 blockDim_y(8, 32);
    dim3 gridDim_y((cols + blockDim_y.x - 1) / blockDim_y.x, (rows + blockDim_y.y - 1) / blockDim_y.y);

    size_t x_shared = (blockDim_x.x + 2 * center) * blockDim_x.y * sizeof(unsigned char);
    size_t y_shared = (blockDim_y.y + 2 * center) * blockDim_y.x * sizeof(float);

    // Launch horizontal Gaussian kernel
    hipEventRecord(start);
    gaussian_smooth_x_kernel<<<gridDim_x, blockDim_x, x_shared>>>(d_image, d_temp, rows, cols, center);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&gpu_time, start, stop);
    printf("gaussian_smooth_x_kernel execution time: %.2f ms\n", gpu_time);
    
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "gaussian_smooth_x_kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        hipFree(d_image);
        hipFree(d_temp);
        hipFree(d_smoothed);
        exit(1);
    }

    // Launch vertical Gaussian kernel
    hipEventRecord(start);
    gaussian_smooth_y_kernel<<<gridDim_y, blockDim_y, y_shared>>>(d_temp, d_smoothed, rows, cols, center);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&gpu_time, start, stop);
    printf("gaussian_smooth_y_kernel execution time: %.2f ms\n", gpu_time);
    
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "gaussian_smooth_y_kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        hipFree(d_image);
        hipFree(d_temp);
        hipFree(d_smoothed);
        exit(1);
    }

    // Allocate and copy result back to host
    *smoothedim = (short int *)malloc(smoothSize);
    if (*smoothedim == NULL) {
        fprintf(stderr, "Error allocating the smoothedim array.\n");
        hipFree(d_image);
        hipFree(d_temp);
        hipFree(d_smoothed);
        exit(1);
    }

    hipEventRecord(start);
    cudaStatus = hipMemcpy(*smoothedim, d_smoothed, smoothSize, hipMemcpyDeviceToHost);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&gpu_time, start, stop);
    printf("hipMemcpy from device time: %.2f ms\n", gpu_time);
    
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy smoothedim failed: %s\n", hipGetErrorString(cudaStatus));
        free(*smoothedim);
        hipFree(d_image);
        hipFree(d_temp);
        hipFree(d_smoothed);
        exit(1);
    }

    // Cleanup
    hipFree(d_image);
    hipFree(d_temp);
    hipFree(d_smoothed);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize failed: %s\n", hipGetErrorString(cudaStatus));
        exit(1);
    }

    double total_end_time = get_time_ms();
    printf("Total GPU function time: %.2f ms\n", total_end_time - total_start_time);
}

// GPU-accelerated Canny edge detection (currently only Gaussian smoothing on GPU)
void cuda_canny(unsigned char *image, int rows, int cols, float sigma,
                float tlow, float thigh, unsigned char **edge, char *fname)
{
    FILE *fpdir = NULL;        /* File to write the gradient image to.     */
    unsigned char *nms;        /* Points that are local maximal magnitude. */
    short int *smoothedim,     /* The image after gaussian smoothing.      */
        *delta_x,              /* The first devivative image, x-direction. */
        *delta_y,              /* The first derivative image, y-direction. */
        *magnitude;            /* The magnitude of the gadient image.      */
    float *dir_radians = NULL; /* Gradient direction image.                */

    // Variables for timing
    double start_time, end_time, step_time;
    double total_time = 0.0;

    /****************************************************************************
     * Perform gaussian smoothing on the image using GPU.
     ****************************************************************************/
    if (VERBOSE)
        printf("Smoothing the image using a gaussian kernel on GPU.\n");

    start_time = get_time_ms();
    cuda_gaussian_smooth(image, rows, cols, sigma, &smoothedim);
    end_time = get_time_ms();
    step_time = end_time - start_time;
    total_time += step_time;
    printf("Gaussian smoothing time: %.2f ms\n", step_time);

    /****************************************************************************
     * Compute the first derivative in the x and y directions (CPU).
     ****************************************************************************/
    if (VERBOSE)
        printf("Computing the X and Y first derivatives.\n");

    start_time = get_time_ms();
    derivative_x_y(smoothedim, rows, cols, &delta_x, &delta_y);
    end_time = get_time_ms();
    step_time = end_time - start_time;
    total_time += step_time;
    printf("X and Y derivatives computation time: %.2f ms\n", step_time);

    /****************************************************************************
     * Direction calculation for edge quality figure of merit (CPU).
     ****************************************************************************/
    if (fname != NULL)
    {
        start_time = get_time_ms();
        radian_direction(delta_x, delta_y, rows, cols, &dir_radians, -1, -1);

        if ((fpdir = fopen(fname, "wb")) == NULL)
        {
            fprintf(stderr, "Error opening the file %s for writing.\n", fname);
            exit(1);
        }
        fwrite(dir_radians, sizeof(float), rows * cols, fpdir);
        fclose(fpdir);
        end_time = get_time_ms();
        step_time = end_time - start_time;
        total_time += step_time;
        printf("Direction calculation time: %.2f ms\n", step_time);

        free(dir_radians);
    }

    /****************************************************************************
     * Compute the magnitude of the gradient (CPU).
     ****************************************************************************/
    if (VERBOSE)
        printf("Computing the magnitude of the gradient.\n");

    start_time = get_time_ms();
    magnitude_x_y(delta_x, delta_y, rows, cols, &magnitude);
    end_time = get_time_ms();
    step_time = end_time - start_time;
    total_time += step_time;
    printf("Gradient magnitude computation time: %.2f ms\n", step_time);

    /****************************************************************************
     * Perform non-maximal suppression (CPU).
     ****************************************************************************/
    if (VERBOSE)
        printf("Doing the non-maximal suppression.\n");

    start_time = get_time_ms();
    if ((nms = (unsigned char *)malloc(rows * cols * sizeof(unsigned char))) == NULL)
    {
        fprintf(stderr, "Error allocating the nms image.\n");
        exit(1);
    }
    non_max_supp(magnitude, delta_x, delta_y, rows, cols, nms);
    end_time = get_time_ms();
    step_time = end_time - start_time;
    total_time += step_time;
    printf("Non-maximal suppression time: %.2f ms\n", step_time);

    /****************************************************************************
     * Use hysteresis to mark the edge pixels (CPU).
     ****************************************************************************/
    if (VERBOSE)
        printf("Doing hysteresis thresholding.\n");

    start_time = get_time_ms();
    if ((*edge = (unsigned char *)malloc(rows * cols * sizeof(unsigned char))) == NULL)
    {
        fprintf(stderr, "Error allocating the edge image.\n");
        exit(1);
    }
    apply_hysteresis(magnitude, nms, rows, cols, tlow, thigh, *edge);
    end_time = get_time_ms();
    step_time = end_time - start_time;
    total_time += step_time;
    printf("Hysteresis thresholding time: %.2f ms\n", step_time);

    // Print total time
    printf("Total Canny edge detection time: %.2f ms\n", total_time);

    /****************************************************************************
     * Free allocated memory.
     ****************************************************************************/
    free(smoothedim);
    free(delta_x);
    free(delta_y);
    free(magnitude);
    free(nms);
}
