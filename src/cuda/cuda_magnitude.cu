#include "hip/hip_runtime.h"
#include "cuda_magnitude.cuh"

__global__ void magnitude_kernel(short int *delta_x, short int *delta_y, short int *magnitude, int rows, int cols) {
    int c = blockIdx.x * blockDim.x + threadIdx.x;
    int r = blockIdx.y * blockDim.y + threadIdx.y;
    if (r < rows && c < cols) {
        int pos = r * cols + c;
        int dx = (int)delta_x[pos];
        int dy = (int)delta_y[pos];
        int sum = dx * dx + dy * dy;
        magnitude[pos] = (short)(0.5f + sqrtf((float)sum));
    }
}

void cuda_magnitude_x_y(short int *delta_x, short int *delta_y, int rows, int cols, short int **magnitude) {
    hipError_t cudaStatus;
    hipEvent_t start, stop;
    float gpu_time;

    // Allocate host memory for magnitude
    *magnitude = (short int *)malloc(rows * cols * sizeof(short int));
    if (*magnitude == NULL) {
        fprintf(stderr, "Error allocating the magnitude image on host.\n");
        return;
    }

    // Create CUDA events
    cudaStatus = hipEventCreate(&start);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipEventCreate for start failed: %s\n", hipGetErrorString(cudaStatus));
        return;
    }
    cudaStatus = hipEventCreate(&stop);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipEventCreate for stop failed: %s\n", hipGetErrorString(cudaStatus));
        hipEventDestroy(start);
        return;
    }

    size_t size = rows * cols * sizeof(short int);
    short int *d_delta_x = NULL;
    short int *d_delta_y = NULL;
    short int *d_magnitude = NULL;

    // Allocate device memory for delta_x
    hipEventRecord(start);
    cudaStatus = hipMalloc((void**)&d_delta_x, size);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&gpu_time, start, stop);
    printf("hipMalloc d_delta_x: %.2f ms\n", gpu_time);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc d_delta_x failed: %s\n", hipGetErrorString(cudaStatus));
        hipEventDestroy(start);
        hipEventDestroy(stop);
        return;
    }

    // Allocate device memory for delta_y
    hipEventRecord(start);
    cudaStatus = hipMalloc((void**)&d_delta_y, size);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&gpu_time, start, stop);
    printf("hipMalloc d_delta_y: %.2f ms\n", gpu_time);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc d_delta_y failed: %s\n", hipGetErrorString(cudaStatus));
        hipFree(d_delta_x);
        hipEventDestroy(start);
        hipEventDestroy(stop);
        return;
    }

    // Allocate device memory for magnitude
    hipEventRecord(start);
    cudaStatus = hipMalloc((void**)&d_magnitude, size);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&gpu_time, start, stop);
    printf("hipMalloc d_magnitude: %.2f ms\n", gpu_time);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc d_magnitude failed: %s\n", hipGetErrorString(cudaStatus));
        hipFree(d_delta_x);
        hipFree(d_delta_y);
        hipEventDestroy(start);
        hipEventDestroy(stop);
        return;
    }

    // Copy delta_x and delta_y from host to device
    hipEventRecord(start);
    cudaStatus = hipMemcpy(d_delta_x, delta_x, size, hipMemcpyHostToDevice);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&gpu_time, start, stop);
    printf("hipMemcpy HtoD delta_x: %.2f ms\n", gpu_time);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy delta_x HtoD failed: %s\n", hipGetErrorString(cudaStatus));
        hipFree(d_delta_x);
        hipFree(d_delta_y);
        hipFree(d_magnitude);
        hipEventDestroy(start);
        hipEventDestroy(stop);
        return;
    }

    hipEventRecord(start);
    cudaStatus = hipMemcpy(d_delta_y, delta_y, size, hipMemcpyHostToDevice);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&gpu_time, start, stop);
    printf("hipMemcpy HtoD delta_y: %.2f ms\n", gpu_time);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy delta_y HtoD failed: %s\n", hipGetErrorString(cudaStatus));
        hipFree(d_delta_x);
        hipFree(d_delta_y);
        hipFree(d_magnitude);
        hipEventDestroy(start);
        hipEventDestroy(stop);
        return;
    }

    // Configure kernel launch parameters
    dim3 block(16, 16);
    dim3 grid((cols + block.x - 1) / block.x, (rows + block.y - 1) / block.y);

    // Launch kernel
    hipEventRecord(start);
    magnitude_kernel<<<grid, block>>>(d_delta_x, d_delta_y, d_magnitude, rows, cols);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&gpu_time, start, stop);
    printf("Kernel execution: %.2f ms\n", gpu_time);

    // Check for kernel launch errors
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        hipFree(d_delta_x);
        hipFree(d_delta_y);
        hipFree(d_magnitude);
        hipEventDestroy(start);
        hipEventDestroy(stop);
        return;
    }

    // Check for kernel execution errors
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Kernel execution failed: %s\n", hipGetErrorString(cudaStatus));
        hipFree(d_delta_x);
        hipFree(d_delta_y);
        hipFree(d_magnitude);
        hipEventDestroy(start);
        hipEventDestroy(stop);
        return;
    }

    // Copy result from device to host
    hipEventRecord(start);
    cudaStatus = hipMemcpy(*magnitude, d_magnitude, size, hipMemcpyDeviceToHost);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&gpu_time, start, stop);
    printf("hipMemcpy DtoH magnitude: %.2f ms\n", gpu_time);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy magnitude DtoH failed: %s\n", hipGetErrorString(cudaStatus));
        hipFree(d_delta_x);
        hipFree(d_delta_y);
        hipFree(d_magnitude);
        hipEventDestroy(start);
        hipEventDestroy(stop);
        return;
    }

    // Cleanup
    hipFree(d_delta_x);
    hipFree(d_delta_y);
    hipFree(d_magnitude);
    hipEventDestroy(start);
    hipEventDestroy(stop);
}