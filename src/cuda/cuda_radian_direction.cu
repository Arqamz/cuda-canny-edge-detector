#include "hip/hip_runtime.h"
#include "cuda_radian_direction.cuh"

__global__ void compute_radian_direction_kernel(const short int* delta_x, const short int* delta_y, float* dir_radians, int rows, int cols) {
    int c = blockIdx.x * blockDim.x + threadIdx.x;
    int r = blockIdx.y * blockDim.y + threadIdx.y;
    if (r >= rows || c >= cols) return;

    int pos = r * cols + c;

    float dx = static_cast<float>(delta_x[pos]);
    float dy = static_cast<float>(delta_y[pos]);

    // Apply sign based on constant tags
    if (c_xdirtag == 1) dx = -dx;
    if (c_ydirtag == -1) dy = -dy;

    if (dx == 0.0f && dy == 0.0f) {
        dir_radians[pos] = 0.0f;
        return;
    }

    float ang = atan2f(dy, dx);
    if (ang < 0.0f) {
        ang += 2.0f * HIP_PI_F;
    }

    dir_radians[pos] = ang;
}

void cuda_radian_direction(short int *delta_x, short int *delta_y, int rows, int cols, float **dir_radians, int xdirtag, int ydirtag) {
    hipError_t cudaStatus;
    hipEvent_t start, stop;
    float gpu_time = 0.0f;

    // Create CUDA events for timing
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Allocate host memory for the direction image
    int num_pixels = rows * cols;
    float *dirim = (float *)malloc(num_pixels * sizeof(float));
    if (dirim == NULL) {
        fprintf(stderr, "Error allocating the gradient direction image.\n");
        hipEventDestroy(start);
        hipEventDestroy(stop);
        return;
    }
    *dir_radians = dirim;

    // Allocate device memory for delta_x, delta_y, and dir_radians
    short int *d_delta_x = NULL;
    short int *d_delta_y = NULL;
    float *d_dir_radians = NULL;
    size_t delta_size = num_pixels * sizeof(short int);
    size_t dir_size = num_pixels * sizeof(float);

    // Allocate d_delta_x
    hipEventRecord(start);
    cudaStatus = hipMalloc((void**)&d_delta_x, delta_size);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&gpu_time, start, stop);
    printf("hipMalloc d_delta_x: %.2f ms\n", gpu_time);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc d_delta_x failed: %s\n", hipGetErrorString(cudaStatus));
        hipEventDestroy(start);
        hipEventDestroy(stop);
        free(dirim);
        return;
    }

    // Allocate d_delta_y
    hipEventRecord(start);
    cudaStatus = hipMalloc((void**)&d_delta_y, delta_size);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&gpu_time, start, stop);
    printf("hipMalloc d_delta_y: %.2f ms\n", gpu_time);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc d_delta_y failed: %s\n", hipGetErrorString(cudaStatus));
        hipFree(d_delta_x);
        hipEventDestroy(start);
        hipEventDestroy(stop);
        free(dirim);
        return;
    }

    // Allocate d_dir_radians
    hipEventRecord(start);
    cudaStatus = hipMalloc((void**)&d_dir_radians, dir_size);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&gpu_time, start, stop);
    printf("hipMalloc d_dir_radians: %.2f ms\n", gpu_time);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc d_dir_radians failed: %s\n", hipGetErrorString(cudaStatus));
        hipFree(d_delta_x);
        hipFree(d_delta_y);
        hipEventDestroy(start);
        hipEventDestroy(stop);
        free(dirim);
        return;
    }

    // Copy xdirtag and ydirtag to constant memory
    hipEventRecord(start);
    cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(c_xdirtag), &xdirtag, sizeof(int));
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&gpu_time, start, stop);
    printf("hipMemcpyToSymbol c_xdirtag: %.2f ms\n", gpu_time);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpyToSymbol c_xdirtag failed: %s\n", hipGetErrorString(cudaStatus));
        hipFree(d_delta_x);
        hipFree(d_delta_y);
        hipFree(d_dir_radians);
        hipEventDestroy(start);
        hipEventDestroy(stop);
        free(dirim);
        return;
    }

    hipEventRecord(start);
    cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(c_ydirtag), &ydirtag, sizeof(int));
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&gpu_time, start, stop);
    printf("hipMemcpyToSymbol c_ydirtag: %.2f ms\n", gpu_time);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpyToSymbol c_ydirtag failed: %s\n", hipGetErrorString(cudaStatus));
        hipFree(d_delta_x);
        hipFree(d_delta_y);
        hipFree(d_dir_radians);
        hipEventDestroy(start);
        hipEventDestroy(stop);
        free(dirim);
        return;
    }

    // Copy input data from host to device
    hipEventRecord(start);
    cudaStatus = hipMemcpy(d_delta_x, delta_x, delta_size, hipMemcpyHostToDevice);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&gpu_time, start, stop);
    printf("hipMemcpy delta_x to device: %.2f ms\n", gpu_time);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy delta_x failed: %s\n", hipGetErrorString(cudaStatus));
        hipFree(d_delta_x);
        hipFree(d_delta_y);
        hipFree(d_dir_radians);
        hipEventDestroy(start);
        hipEventDestroy(stop);
        free(dirim);
        return;
    }

    hipEventRecord(start);
    cudaStatus = hipMemcpy(d_delta_y, delta_y, delta_size, hipMemcpyHostToDevice);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&gpu_time, start, stop);
    printf("hipMemcpy delta_y to device: %.2f ms\n", gpu_time);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy delta_y failed: %s\n", hipGetErrorString(cudaStatus));
        hipFree(d_delta_x);
        hipFree(d_delta_y);
        hipFree(d_dir_radians);
        hipEventDestroy(start);
        hipEventDestroy(stop);
        free(dirim);
        return;
    }

    // Launch kernel
    dim3 block(16, 16);
    dim3 grid((cols + block.x - 1) / block.x, (rows + block.y - 1) / block.y);

    hipEventRecord(start);
    compute_radian_direction_kernel<<<grid, block>>>(d_delta_x, d_delta_y, d_dir_radians, rows, cols);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&gpu_time, start, stop);
    printf("Kernel execution time: %.2f ms\n", gpu_time);

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        hipFree(d_delta_x);
        hipFree(d_delta_y);
        hipFree(d_dir_radians);
        hipEventDestroy(start);
        hipEventDestroy(stop);
        free(dirim);
        return;
    }

    // Copy output from device to host
    hipEventRecord(start);
    cudaStatus = hipMemcpy(dirim, d_dir_radians, dir_size, hipMemcpyDeviceToHost);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&gpu_time, start, stop);
    printf("hipMemcpy dir_radians to host: %.2f ms\n", gpu_time);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy dir_radians failed: %s\n", hipGetErrorString(cudaStatus));
        hipFree(d_delta_x);
        hipFree(d_delta_y);
        hipFree(d_dir_radians);
        hipEventDestroy(start);
        hipEventDestroy(stop);
        free(dirim);
        return;
    }

    // Cleanup
    hipFree(d_delta_x);
    hipFree(d_delta_y);
    hipFree(d_dir_radians);
    hipEventDestroy(start);
    hipEventDestroy(stop);
}