#include "hip/hip_runtime.h"
#include "cuda_derivative.cuh"

// X derivative kernel using shared memory for row data
__global__ void compute_delta_x_kernel(const short int *smoothedim, short int *delta_x, int rows, int cols) {
    extern __shared__ short int row_data[];
    int row = blockIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < rows) {
        // Load entire row into shared memory
        for (int c = threadIdx.x; c < cols; c += blockDim.x) {
            row_data[c] = smoothedim[row * cols + c];
        }
        __syncthreads();

        if (col < cols) {
            if (col == 0) {
                delta_x[row * cols + col] = row_data[col + 1] - row_data[col];
            } else if (col == cols - 1) {
                delta_x[row * cols + col] = row_data[col] - row_data[col - 1];
            } else {
                delta_x[row * cols + col] = row_data[col + 1] - row_data[col - 1];
            }
        }
    }
}

// Y derivative kernel using column-wise shared memory tiles with halo
__global__ void compute_delta_y_kernel(const short int *smoothedim, short int *delta_y, int rows, int cols) {
    extern __shared__ short int shared_col[];
    int col = blockIdx.x;
    int row_in_block = threadIdx.y;
    int tile_start_row = blockIdx.y * blockDim.y;

    // Load main tile elements
    int load_row = tile_start_row + row_in_block;
    if (load_row < rows && col < cols) {
        shared_col[row_in_block + 1] = smoothedim[load_row * cols + col];
    }

    // Load halo above
    if (row_in_block == 0) {
        int halo_row = tile_start_row - 1;
        if (halo_row < 0) halo_row = 0;
        shared_col[0] = smoothedim[halo_row * cols + col];
    }

    // Load halo below
    if (row_in_block == blockDim.y - 1) {
        int halo_row = tile_start_row + blockDim.y;
        if (halo_row >= rows) halo_row = rows - 1;
        shared_col[blockDim.y + 1] = smoothedim[halo_row * cols + col];
    }

    __syncthreads();

    int row = tile_start_row + row_in_block;
    if (row >= rows || col >= cols) return;

    if (row == 0) {
        delta_y[row * cols + col] = shared_col[1 + 1] - shared_col[1];
    } else if (row == rows - 1) {
        delta_y[row * cols + col] = shared_col[blockDim.y + 1] - shared_col[blockDim.y];
    } else {
        delta_y[row * cols + col] = shared_col[row_in_block + 2] - shared_col[row_in_block];
    }
}

void cuda_derivative_x_y(short int *smoothedim, int rows, int cols, short int **delta_x, short int **delta_y) {
    hipError_t cudaStatus;
    hipEvent_t start, stop;
    float gpu_time = 0.0f;

    short int *d_smoothed = NULL, *d_delta_x = NULL, *d_delta_y = NULL;
    size_t size = rows * cols * sizeof(short int);

    // Initialize CUDA events
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Allocate device memory
    hipEventRecord(start);
    cudaStatus = hipMalloc((void**)&d_smoothed, size);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&gpu_time, start, stop);
    printf("hipMalloc d_smoothed: %.2f ms\n", gpu_time);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc d_smoothed failed: %s\n", hipGetErrorString(cudaStatus));
        hipEventDestroy(start);
        hipEventDestroy(stop);
        return;
    }

    hipEventRecord(start);
    cudaStatus = hipMalloc((void**)&d_delta_x, size);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&gpu_time, start, stop);
    printf("hipMalloc d_delta_x: %.2f ms\n", gpu_time);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc d_delta_x failed: %s\n", hipGetErrorString(cudaStatus));
        hipFree(d_smoothed);
        hipEventDestroy(start);
        hipEventDestroy(stop);
        return;
    }

    hipEventRecord(start);
    cudaStatus = hipMalloc((void**)&d_delta_y, size);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&gpu_time, start, stop);
    printf("hipMalloc d_delta_y: %.2f ms\n", gpu_time);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc d_delta_y failed: %s\n", hipGetErrorString(cudaStatus));
        hipFree(d_smoothed);
        hipFree(d_delta_x);
        hipEventDestroy(start);
        hipEventDestroy(stop);
        return;
    }

    // Copy input to device
    hipEventRecord(start);
    cudaStatus = hipMemcpy(d_smoothed, smoothedim, size, hipMemcpyHostToDevice);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&gpu_time, start, stop);
    printf("hipMemcpy to device: %.2f ms\n", gpu_time);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy d_smoothed failed: %s\n", hipGetErrorString(cudaStatus));
        hipFree(d_smoothed);
        hipFree(d_delta_x);
        hipFree(d_delta_y);
        hipEventDestroy(start);
        hipEventDestroy(stop);
        return;
    }

    // Launch delta_x kernel
    dim3 block_x(TILE_WIDTH, 1);
    dim3 grid_x((cols + TILE_WIDTH - 1) / TILE_WIDTH, rows);
    size_t shared_x = cols * sizeof(short);

    hipEventRecord(start);
    compute_delta_x_kernel<<<grid_x, block_x, shared_x>>>(d_smoothed, d_delta_x, rows, cols);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&gpu_time, start, stop);
    printf("compute_delta_x_kernel: %.2f ms\n", gpu_time);
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "delta_x kernel failed: %s\n", hipGetErrorString(cudaStatus));
        hipFree(d_smoothed);
        hipFree(d_delta_x);
        hipFree(d_delta_y);
        hipEventDestroy(start);
        hipEventDestroy(stop);
        return;
    }

    // Launch delta_y kernel
    dim3 block_y(1, TILE_HEIGHT);
    dim3 grid_y(cols, (rows + TILE_HEIGHT - 1) / TILE_HEIGHT);
    size_t shared_y = (TILE_HEIGHT + 2) * sizeof(short);

    hipEventRecord(start);
    compute_delta_y_kernel<<<grid_y, block_y, shared_y>>>(d_smoothed, d_delta_y, rows, cols);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&gpu_time, start, stop);
    printf("compute_delta_y_kernel: %.2f ms\n", gpu_time);
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "delta_y kernel failed: %s\n", hipGetErrorString(cudaStatus));
        hipFree(d_smoothed);
        hipFree(d_delta_x);
        hipFree(d_delta_y);
        hipEventDestroy(start);
        hipEventDestroy(stop);
        return;
    }

    // Allocate host memory
    *delta_x = (short int*)malloc(size);
    *delta_y = (short int*)malloc(size);
    if (!*delta_x || !*delta_y) {
        fprintf(stderr, "Host memory allocation failed\n");
        hipFree(d_smoothed);
        hipFree(d_delta_x);
        hipFree(d_delta_y);
        hipEventDestroy(start);
        hipEventDestroy(stop);
        return;
    }

    // Copy results back
    hipEventRecord(start);
    cudaStatus = hipMemcpy(*delta_x, d_delta_x, size, hipMemcpyDeviceToHost);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&gpu_time, start, stop);
    printf("hipMemcpy delta_x: %.2f ms\n", gpu_time);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy delta_x failed: %s\n", hipGetErrorString(cudaStatus));
        free(*delta_x);
        free(*delta_y);
        hipFree(d_smoothed);
        hipFree(d_delta_x);
        hipFree(d_delta_y);
        hipEventDestroy(start);
        hipEventDestroy(stop);
        return;
    }

    hipEventRecord(start);
    cudaStatus = hipMemcpy(*delta_y, d_delta_y, size, hipMemcpyDeviceToHost);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&gpu_time, start, stop);
    printf("hipMemcpy delta_y: %.2f ms\n", gpu_time);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy delta_y failed: %s\n", hipGetErrorString(cudaStatus));
        free(*delta_x);
        free(*delta_y);
        hipFree(d_smoothed);
        hipFree(d_delta_x);
        hipFree(d_delta_y);
        hipEventDestroy(start);
        hipEventDestroy(stop);
        return;
    }

    // Cleanup
    hipFree(d_smoothed);
    hipFree(d_delta_x);
    hipFree(d_delta_y);
    hipEventDestroy(start);
    hipEventDestroy(stop);
}