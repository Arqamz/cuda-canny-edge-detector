#include "hip/hip_runtime.h"
#include "cuda_hysteresis.cuh"

// Constants for directional offsets
__constant__ int c_dx[8] = {1, 1, 0, -1, -1, -1, 0, 1};
__constant__ int c_dy[8] = {0, 1, 1, 1, 0, -1, -1, -1};

// Kernel for first step: marking initial edges above high threshold
__global__ void mark_initial_edges_kernel(short* mag, unsigned char* edge, int rows, int cols, int highthreshold) {
    int c = blockIdx.x * blockDim.x + threadIdx.x;
    int r = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (r < rows && c < cols) {
        int pos = r * cols + c;
        if (edge[pos] == POSSIBLE_EDGE && mag[pos] >= highthreshold) {
            edge[pos] = EDGE;
        }
    }
}

// Kernel for edge propagation (iterative)
__global__ void propagate_edges_kernel(unsigned char* edge, short* mag, int rows, int cols, int lowthreshold, int* changed) {
    int c = blockIdx.x * blockDim.x + threadIdx.x;
    int r = blockIdx.y * blockDim.y + threadIdx.y;
    
    __shared__ int block_changed;
    
    if (threadIdx.x == 0 && threadIdx.y == 0) {
        block_changed = 0;
    }
    __syncthreads();
    
    if (r < rows && c < cols) {
        int pos = r * cols + c;
        
        // Only POSSIBLE_EDGE pixels need to be checked
        if (edge[pos] == POSSIBLE_EDGE) {
            // Check all 8 neighbors
            for (int i = 0; i < 8; i++) {
                int nr = r + c_dy[i];
                int nc = c + c_dx[i];
                
                // Boundary check
                if (nr < 0 || nr >= rows || nc < 0 || nc >= cols) {
                    continue;
                }
                
                int npos = nr * cols + nc;
                
                // If neighbor is an EDGE and this pixel is above low threshold
                if (edge[npos] == EDGE && mag[pos] >= lowthreshold) {
                    edge[pos] = EDGE;
                    atomicExch(&block_changed, 1);
                    break;
                }
            }
        }
    }
    
    __syncthreads();
    
    // Only one thread per block updates the global flag
    if (threadIdx.x == 0 && threadIdx.y == 0 && block_changed) {
        atomicExch(changed, 1);
    }
}

// Kernel for final cleanup (set all non-EDGE to NOEDGE)
__global__ void cleanup_edges_kernel(unsigned char* edge, int rows, int cols) {
    int c = blockIdx.x * blockDim.x + threadIdx.x;
    int r = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (r < rows && c < cols) {
        int pos = r * cols + c;
        if (edge[pos] != EDGE) {
            edge[pos] = NOEDGE;
        }
    }
}

// Initialize edge map (border cleaning and marking possible edges)
__global__ void initialize_edge_map_kernel(unsigned char* nms, unsigned char* edge, int rows, int cols) {
    int c = blockIdx.x * blockDim.x + threadIdx.x;
    int r = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (r < rows && c < cols) {
        int pos = r * cols + c;
        
        // Mark as possible edge or not
        if (nms[pos] == POSSIBLE_EDGE) {
            edge[pos] = POSSIBLE_EDGE;
        } else {
            edge[pos] = NOEDGE;
        }
        
        // Clear borders
        if (r == 0 || r == rows-1 || c == 0 || c == cols-1) {
            edge[pos] = NOEDGE;
        }
    }
}

// Kernel to compute histogram
__global__ void compute_histogram_kernel(short* mag, unsigned char* edge, int rows, int cols, int* hist) {
    int c = blockIdx.x * blockDim.x + threadIdx.x;
    int r = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (r < rows && c < cols) {
        int pos = r * cols + c;
        if (edge[pos] == POSSIBLE_EDGE) {
            atomicAdd(&hist[mag[pos]], 1);
        }
    }
}

// Main CUDA implementation of hysteresis
hipError_t cuda_apply_hysteresis(short* mag, unsigned char* nms, int rows, int cols, 
                                 float tlow, float thigh, unsigned char* edge) {
    hipError_t cudaStatus;
    float gpu_time = 0.0f;
    hipEvent_t start, stop;
    
    // Initialize CUDA events for timing
    CHECK_CUDA_ERROR(hipEventCreate(&start));
    CHECK_CUDA_ERROR(hipEventCreate(&stop));
    
    // Allocate device memory
    short* d_mag = nullptr;
    unsigned char* d_nms = nullptr;
    unsigned char* d_edge = nullptr;
    int* d_hist = nullptr;
    int* d_changed = nullptr;
    
    size_t image_size = rows * cols * sizeof(unsigned char);
    size_t mag_size = rows * cols * sizeof(short);
    size_t hist_size = 32768 * sizeof(int);
    
    printf("Allocating device memory...\n");
    
    // Allocate and copy magnitude data
    hipEventRecord(start);
    CHECK_CUDA_ERROR(hipMalloc((void**)&d_mag, mag_size));
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&gpu_time, start, stop);
    printf("hipMalloc magnitude: %.2f ms\n", gpu_time);
    
    hipEventRecord(start);
    CHECK_CUDA_ERROR(hipMemcpy(d_mag, mag, mag_size, hipMemcpyHostToDevice));
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&gpu_time, start, stop);
    printf("hipMemcpy HtoD magnitude: %.2f ms\n", gpu_time);
    
    // Allocate and copy non-maximal suppression data
    hipEventRecord(start);
    CHECK_CUDA_ERROR(hipMalloc((void**)&d_nms, image_size));
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&gpu_time, start, stop);
    printf("hipMalloc nms: %.2f ms\n", gpu_time);
    
    hipEventRecord(start);
    CHECK_CUDA_ERROR(hipMemcpy(d_nms, nms, image_size, hipMemcpyHostToDevice));
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&gpu_time, start, stop);
    printf("hipMemcpy HtoD nms: %.2f ms\n", gpu_time);
    
    // Allocate edge output buffer
    hipEventRecord(start);
    CHECK_CUDA_ERROR(hipMalloc((void**)&d_edge, image_size));
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&gpu_time, start, stop);
    printf("hipMalloc edge: %.2f ms\n", gpu_time);
    
    // Allocate histogram array
    hipEventRecord(start);
    CHECK_CUDA_ERROR(hipMalloc((void**)&d_hist, hist_size));
    CHECK_CUDA_ERROR(hipMemset(d_hist, 0, hist_size));
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&gpu_time, start, stop);
    printf("hipMalloc and clear histogram: %.2f ms\n", gpu_time);
    
    // Allocate changed flag
    hipEventRecord(start);
    CHECK_CUDA_ERROR(hipMalloc((void**)&d_changed, sizeof(int)));
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&gpu_time, start, stop);
    printf("hipMalloc changed flag: %.2f ms\n", gpu_time);
    
    // Configure kernel dimensions
    dim3 blockSize(16, 16);
    dim3 gridSize((cols + blockSize.x - 1) / blockSize.x, 
                  (rows + blockSize.y - 1) / blockSize.y);
    
    // Step 1: Initialize edge map
    hipEventRecord(start);
    initialize_edge_map_kernel<<<gridSize, blockSize>>>(d_nms, d_edge, rows, cols);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&gpu_time, start, stop);
    printf("Initialize edge map kernel: %.2f ms\n", gpu_time);
    CHECK_CUDA_ERROR(hipGetLastError());
    
    // Step 2: Compute histogram
    hipEventRecord(start);
    compute_histogram_kernel<<<gridSize, blockSize>>>(d_mag, d_edge, rows, cols, d_hist);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&gpu_time, start, stop);
    printf("Compute histogram kernel: %.2f ms\n", gpu_time);
    CHECK_CUDA_ERROR(hipGetLastError());
    
    // Copy histogram back to host for threshold calculation
    int* h_hist = new int[32768];
    hipEventRecord(start);
    CHECK_CUDA_ERROR(hipMemcpy(h_hist, d_hist, hist_size, hipMemcpyDeviceToHost));
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&gpu_time, start, stop);
    printf("hipMemcpy DtoH histogram: %.2f ms\n", gpu_time);
    
    // Calculate thresholds on CPU
    int numedges = 0;
    int maximum_mag = 0;
    
    for (int r = 1; r < 32768; r++) {
        if (h_hist[r] != 0) {
            maximum_mag = r;
        }
        numedges += h_hist[r];
    }
    
    int highcount = (int)(numedges * thigh + 0.5);
    
    int r = 1;
    numedges = h_hist[1];
    while ((r < (maximum_mag - 1)) && (numedges < highcount)) {
        r++;
        numedges += h_hist[r];
    }
    int highthreshold = r;
    int lowthreshold = (int)(highthreshold * tlow + 0.5);
    
    printf("Thresholds computed: low = %d, high = %d\n", lowthreshold, highthreshold);
    delete[] h_hist;
    
    // Step 3: Mark initial edges (pixels above high threshold)
    hipEventRecord(start);
    mark_initial_edges_kernel<<<gridSize, blockSize>>>(d_mag, d_edge, rows, cols, highthreshold);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&gpu_time, start, stop);
    printf("Mark initial edges kernel: %.2f ms\n", gpu_time);
    CHECK_CUDA_ERROR(hipGetLastError());
    
    // Step 4: Iterative edge propagation (replaces recursive follow_edges)
    int max_iterations = 100;  // prevent infinite loops
    int iterations = 0;
    
    printf("Starting edge propagation...\n");
    
    do {
        // Reset change flag
        CHECK_CUDA_ERROR(hipMemset(d_changed, 0, sizeof(int)));
        
        // Propagate edges
        hipEventRecord(start);
        propagate_edges_kernel<<<gridSize, blockSize>>>(d_edge, d_mag, rows, cols, lowthreshold, d_changed);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&gpu_time, start, stop);
        if (iterations == 0 || iterations == max_iterations-1) {
            printf("Edge propagation iteration %d: %.2f ms\n", iterations, gpu_time);
        }
        CHECK_CUDA_ERROR(hipGetLastError());
        
        int changed = 0;
        CHECK_CUDA_ERROR(hipMemcpy(&changed, d_changed, sizeof(int), hipMemcpyDeviceToHost));
        
        if (!changed) {
            break;
        }
        
        iterations++;
    } while (iterations < max_iterations);
    
    printf("Edge propagation complete after %d iterations\n", iterations);
    
    // Step 5: Final cleanup (set all non-EDGE pixels to NOEDGE)
    hipEventRecord(start);
    cleanup_edges_kernel<<<gridSize, blockSize>>>(d_edge, rows, cols);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&gpu_time, start, stop);
    printf("Cleanup edges kernel: %.2f ms\n", gpu_time);
    CHECK_CUDA_ERROR(hipGetLastError());
    
    // Copy result back to host
    hipEventRecord(start);
    CHECK_CUDA_ERROR(hipMemcpy(edge, d_edge, image_size, hipMemcpyDeviceToHost));
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&gpu_time, start, stop);
    printf("hipMemcpy DtoH edge result: %.2f ms\n", gpu_time);
    
    // Free device memory
    hipEventRecord(start);
    hipFree(d_mag);
    hipFree(d_nms);
    hipFree(d_edge);
    hipFree(d_hist);
    hipFree(d_changed);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&gpu_time, start, stop);
    printf("Free device memory: %.2f ms\n", gpu_time);
    
    // Destroy CUDA events
    hipEventDestroy(start);
    hipEventDestroy(stop);
    
    return hipSuccess;
}