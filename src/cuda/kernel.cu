#include "hip/hip_runtime.h"
#include "kernel.cuh"

// Basic CUDA kernel that prints "Hello World"
__global__ void helloWorldKernel() {
    printf("Hello World from thread [%d, %d]!\n", threadIdx.x, blockIdx.x);
}

// Host function to launch the kernel
void launchHelloWorldKernel() {
    // Launch the kernel with 1 block of 256 threads
    helloWorldKernel<<<1, 256>>>();
    
    // Synchronize to make sure the kernel completes
    hipDeviceSynchronize();
    
    // Check for any errors
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(error));
    }
}
