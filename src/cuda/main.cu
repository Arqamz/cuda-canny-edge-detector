#include <stdio.h>

#include "kernel.cuh"

int main()
{
    printf("Hello World from CPU!\n");

    launchHelloWorldKernel();
    
    hipError_t cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Kernel execution failed: %s\n", hipGetErrorString(cudaStatus));
        return 1;
    }
    
    printf("GPU computation completed successfully!\n");
    
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed: %s\n", hipGetErrorString(cudaStatus));
        return 1;
    }

    return 0;
}
